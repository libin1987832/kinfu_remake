#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
#include <kfusion/types.hpp>
//#include "pcl/gpu/utils/device/vector_math.hpp"
#include "texture_binder.hpp"
namespace kfusion
{
  namespace device
  {

    __global__ void
		initColorVolumeKernel(PtrStep<uchar4> volume, int VOLUME_X, int VOLUME_Y, int VOLUME_Z)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x < VOLUME_X && y < VOLUME_Y)
      {
        uchar4 *pos = volume.ptr (y) + x;
        int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
        for (int z = 0; z < VOLUME_Z; ++z, pos += z_step)
          *pos = make_uchar4 (0, 0, 0, 0);
      }
    }
  }
}

void
kfusion::device::initColorVolume(PtrStep<uchar4> color_volume, int V_X,int V_Y,int V_Z)
{
  dim3 block (32, 16);
  dim3 grid (1, 1, 1);
  grid.x = divUp(V_X, block.x);
  grid.y = divUp(V_Y, block.y);

  initColorVolumeKernel<<<grid, block>>>(color_volume,V_X,V_Y,V_Z);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace kfusion
{
  namespace device
  {
   texture<float, 2> color_tex(0, hipFilterModePoint, hipAddressModeBorder, cudaCreateChannelDescHalf());

    struct ColorVolumeImpl
    {

      mutable PtrStep<uchar4> color_volume;
	      Aff3f vol2cam;
            Projector proj;
            int2 dists_size;

            float tranc_dist_inv;
			    PtrStepSz<uchar3> colors;
				Intr intr;
            __kf_device__
            void operator()(TsdfVolume& volume) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume.dims.x || y >= volume.dims.y)
                    return;

                //float3 zstep = vol2cam.R * make_float3(0.f, 0.f, volume.voxel_size.z);
                float3 zstep = make_float3(vol2cam.R.data[0].z, vol2cam.R.data[1].z, vol2cam.R.data[2].z) * volume.voxel_size.z;

                float3 vx = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
                float3 vc = vol2cam * vx; //tranform from volume coo frame to camera one

                TsdfVolume::elem_type* vptr = volume.beg(x, y);
                for(int i = 0; i < volume.dims.z; ++i, vc += zstep, vptr = volume.zstep(vptr))
                {
                    float2 coo = proj(vc);

                    //#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
                    // this is actually workaround for kepler. it doesn't return 0.f for texture
                    // fetches for out-of-border coordinates even for cudaaddressmodeborder mode
                    if (coo.x < 0 || coo.y < 0 || coo.x >= dists_size.x || coo.y >= dists_size.y)
                        continue;
                    //#endif
                    float Dp = tex2D(color_tex, coo.x, coo.y);
                    if(Dp == 0 || vc.z <= 0)
                        continue;

                    float sdf = Dp - __fsqrt_rn(dot(vc, vc)); //Dp - norm(v)

                    if (sdf >= -volume.trunc_dist)
                    {
            //if (update)
						{
							uchar4 *ptr = color_volume.ptr(volume.dims.y * i + y) + x;
							uchar3 rgb = colors.ptr (__float2int_rn(coo.y))[__float2int_rn(coo.x)];
							uchar4 volume_rgbw = *ptr;

							int weight_prev = volume_rgbw.w;

							const float Wrk = 1.f;
							float new_x = (volume_rgbw.x * weight_prev + Wrk * rgb.x) / (weight_prev + Wrk);
							float new_y = (volume_rgbw.y * weight_prev + Wrk * rgb.y) / (weight_prev + Wrk);
							float new_z = (volume_rgbw.z * weight_prev + Wrk * rgb.z) / (weight_prev + Wrk);

							int weight_new = weight_prev + 1;

							uchar4 volume_rgbw_new;
							volume_rgbw_new.x = min (255, max (0, __float2int_rn (new_x)));
							volume_rgbw_new.y = min (255, max (0, __float2int_rn (new_y)));
							volume_rgbw_new.z = min (255, max (0, __float2int_rn (new_z)));
							volume_rgbw_new.w = min (volume.max_weight, weight_new);

							*ptr = volume_rgbw_new;
						}
                    }
                }  // for(;;)
            }

    };

    __global__ void
    updateColorVolumeKernel (const ColorVolumeImpl cvi,TsdfVolume volume) {
      cvi (volume);
    }
  }
}

void
kfusion::device::updateColorVolume(const PtrStepSz<ushort>& dists, TsdfVolume& volume, const Aff3f& aff, const Projector& proj, PtrStep<uchar4> color_volume,const PtrStepSz<uchar3>& colors)
{
  ColorVolumeImpl ti;
	ti.dists_size = make_int2(dists.cols, dists.rows);
    ti.vol2cam = aff;
    ti.proj = proj;
    ti.tranc_dist_inv = 1.f/volume.trunc_dist;
	ti.color_volume=color_volume;
	ti.colors=colors;


	 color_tex.filterMode = hipFilterModePoint;
    color_tex.addressMode[0] = hipAddressModeBorder;
    color_tex.addressMode[1] = hipAddressModeBorder;
    color_tex.addressMode[2] = hipAddressModeBorder;
    TextureBinder binder(dists, color_tex, cudaCreateChannelDescHalf()); (void)binder;

   dim3 block(32, 8);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));

  updateColorVolumeKernel<<<grid, block>>>(ti,volume);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace kfusion
{
  namespace device
  {
    __global__ void
		extractColorsKernel(const float3 cell_size, const PtrStep<uchar4> color_volume, const PtrSz<Point> points, uchar4 *colors, int VOLUME_Y)
    {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      if (idx < points.size)
      {
        int3 v;
        float3 p = *(const float3*)(points.data + idx);
        v.x = __float2int_rd (p.x / cell_size.x);        // round to negative infinity
        v.y = __float2int_rd (p.y / cell_size.y);
        v.z = __float2int_rd (p.z / cell_size.z);
		//if (VOLUME_Y * v.z + v.y < 0)
	//		printf("error1:%d %d %d %d", VOLUME_Y * v.z + v.y,v.x,v.y,v.z);
		//else
		//	printf("error2:%d %d %d %d", VOLUME_Y * v.z + v.y, v.x, v.y, v.z);
        uchar4 rgbw = color_volume.ptr (VOLUME_Y * v.z + v.y)[v.x];
        colors[idx] = make_uchar4 (rgbw.z, rgbw.y, rgbw.x, 0); //bgra
      }
    }
  }
}

void
kfusion::device::exctractColors(const PtrStep<uchar4>& color_volume, const float3& volume_size, const PtrSz<Point>& points, uchar4* colors, int VOLUME_X, int VOLUME_Y, int VOLUME_Z)
{
  const int block = 256;
  float3 cell_size = make_float3 (volume_size.x / VOLUME_X, volume_size.y / VOLUME_Y, volume_size.z / VOLUME_Z);
  extractColorsKernel << <divUp(points.size, block), block >> >(cell_size, color_volume, points, colors, VOLUME_Y);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
};
